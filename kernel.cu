
#include <hip/hip_runtime.h>
#include <iostream>
#include <nvtx3/nvToolsExt.h>

// Structure for the vector addition result.
typedef struct
{
    double amount;
    double time;
} VECTOR_ADD_RESULT, *PVECTOR_ADD_RESULT;

/**
 * @brief CUDA kernel for adding two vectors element-wise.
 *
 * This kernel function performs element-wise addition of two vectors and stores the result in a third vector.
 *
 * @param a Pointer to the first input vector.
 * @param b Pointer to the second input vector.
 * @param c Pointer to the output vector.
 * @param n Number of elements in the vectors.
 */
__global__ void VectorAddKernel(double *a, double *b, double *c, int n)
{
    // Calculate our global thread id.
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // We might run with more threads than elements, so we need to make sure we don't do any work outside of our data.
    if (id < n)
        c[id] = a[id] + b[id];
}

/**
 * Calculates the sum of two vectors using CUDA.
 *
 * @param seed The seed value for random number generation.
 * @return A pointer to a VECTOR_ADD_RESULT struct containing the result of the vector addition.
 */
__declspec(dllexport) PVECTOR_ADD_RESULT __cdecl VectorAdd(unsigned int seed)
{
    nvtxRangePushA("VectorAdd");

    nvtxMark("InitializeReturnStruct");
    PVECTOR_ADD_RESULT result = (PVECTOR_ADD_RESULT)malloc(sizeof(VECTOR_ADD_RESULT));
    result->amount = 0;
    result->time = 123.456;

    nvtxMark("SetRandomSeed");
    srand(seed);

    // Number of elements in each vector.
    int n = 500000;

    // Host memory pointers for the input and output vectors.
    double *h_a, *h_b, *h_c;

    // Device memory pointers for the input and output vectors.
    double *d_a, *d_b, *d_c;

    // Number of bytes to allocate.
    size_t bytes = n * sizeof(double);

    // Allocate memory on the host.
    nvtxRangePushA("HostMemAlloc");
    h_a = (double *)malloc(bytes);
    h_b = (double *)malloc(bytes);
    h_c = (double *)malloc(bytes);
    nvtxRangePop();

    // Allocate memory on the device.
    nvtxRangePushA("DeviceMemAlloc");
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
    nvtxRangePop();

    // Initialize vectors on host.
    int i, x;
    for (i = 0; i < n; i++)
    {
        x = rand();
        h_a[i] = sin(i) * sin(x) + x;
        h_b[i] = cos(i) * cos(i) - x;
    }

    nvtxRangePushA("CopyVectorsToDevice");
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
    nvtxRangePop();

    // Calculate the number of thread blocks.
    int blockSize, gridSize;
    blockSize = 5000;
    gridSize = (int)ceil((float)n / blockSize);

    nvtxRangePushA("KernelExecution");
    VectorAddKernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    nvtxRangePop();

    nvtxRangePushA("CopyResultBackToHost");
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    nvtxRangePop();

    // Consolidate the result.
    double sum = 0;
    for (i = 0; i < n; i++)
        sum += h_c[i] + 1;

    // Set the result on the result struct.
    result->amount = 2 + sum / n;

    nvtxMark("FreeBothDeviceAndHostMemory");
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    nvtxRangePop();

    return result;
}
